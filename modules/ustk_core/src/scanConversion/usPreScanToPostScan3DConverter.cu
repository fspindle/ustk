#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * This file is part of the ustk software.
 * Copyright (C) 2016 - 2017 by Inria. All rights reserved.
 *
 * This software is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * ("GPL") version 2 as published by the Free Software Foundation.
 * See the file LICENSE.txt at the root directory of this source
 * distribution for additional information about the GNU GPL.
 *
 * For using ustk with software that can not be combined with the GNU
 * GPL, please contact Inria about acquiring a ViSP Professional
 * Edition License.
 *
 * This software was developed at:
 * Inria Rennes - Bretagne Atlantique
 * Campus Universitaire de Beaulieu
 * 35042 Rennes Cedex
 * France
 *
 * If you have questions regarding the use of this file, please contact
 * Inria at ustk@inria.fr
 *
 * This file is provided AS IS with NO WARRANTY OF ANY KIND, INCLUDING THE
 * WARRANTY OF DESIGN, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE.
 *
 * Authors:
 * Jason Chevrie
 *
 *****************************************************************************/

#include <visp3/ustk_core/usPreScanToPostScan3DConverter.h>

#ifdef USTK_HAVE_CUDA
void usPreScanToPostScan3DConverter::GPUDirectConversion()
{
    int X = m_VpreScan.getWidth();
    int Y = m_VpreScan.getHeight();
    int Z = m_VpreScan.getNumberOfFrames();
    
    double xmax;
    double ymin;
    double ymax;
    double zmax;
    
    usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord(0.0, X, Z, &ymin, NULL, NULL);
    usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, X / 2.0, Z / 2.0, &ymax, NULL, NULL);
    usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, (double)X, Z / 2.0, NULL, &xmax, NULL);
    usPreScanToPostScan3DConverter::convertPreScanCoordToPostScanCoord((double)Y, X / 2.0, Z, NULL, NULL, &zmax);
    
    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks(m_nbX/threadsPerBlock.x, m_nbY/threadsPerBlock.y, m_nbZ/threadsPerBlock.z);
    usPreScanToPostScan3DConverter::kernelPostScanVoxelDirect<<<numBlocks,threadsPerBlock>>>(dataPost, dataPre, X, Y, Z, xmax, ymin, zmax);
}
    
__global__ void usPreScanToPostScan3DConverter::kernelPostScanVoxelDirect(unsigned char *dataPost, const unsigned char *dataPre, int X, int Y, int Z, double xmax_post, double ymin_post, double zmax_post)
{
    uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
    uint z = (blockIdx.z * blockDim.z) + threadIdx.z;
    
    if(x >= m_nbX  || y >= m_nbY || z >= m_nbZ) return;
  
    unsigned int nbXY = m_nbX * m_nbY;
    unsigned int XY = X * Y;

    double xx = m_resolution * x - xmax_post;
    double yy = ymin_post + m_resolution * y;
    double zz = m_resolution * z - zmax_post;

    double i, j, k;
    usPreScanToPostScan3DConverter::convertPostScanCoordToPreScanCoord(yy, xx, zz, &j, &i, &k, m_SweepInZdirection);
  
    double ii = floor(i);
    double jj = floor(j);
    double kk = floor(k);
  
    if (ii >= 0 && jj >= 0 && kk >= 0 && ii + 1 < X && jj + 1 < Y && kk + 1 < Z) {
              
        double u = i - ii;
        double v = j - jj;
        double w = k - kk;
        double u1 = 1 - u;
        double v1 = 1 - v;
        double w1 = 1 - w;
    
        double v1w1 = v1 * w1;
        double vw1 = v * w1;
        double v1w = v1 * w;
        double vw = v * w;
    
        double W[8] = { u1 * v1w1,
                        u * v1w1,
                        u1 * vw1,
                        u * vw1,
                        u1 * v1w,
                        u * v1w,
                        u1 * vw,
                        u * vw };
        
        double Xjj = X * jj;
        double Xjj1 = X * (jj + 1);
        double XYKK = XY * kk;
        double XYKK1 = XY * (kk + 1);
            
        unsigned int index[8] = { (unsigned int)(ii + Xjj + XYKK),
                                  (unsigned int)(ii + 1 + Xjj + XYKK),
                                  (unsigned int)(ii + Xjj1 + XYKK),
                                  (unsigned int)(ii + 1 + Xjj1 + XYKK),
                                  (unsigned int)(ii + Xjj + XYKK1),
                                  (unsigned int)(ii + 1 + Xjj + XYKK1),
                                  (unsigned int)(ii + Xjj1 + XYKK1),
                                  (unsigned int)(ii + 1 + Xjj1 + XYKK1)};
        
        double val = 0;
        for (int j = 0; j < 8; j++) val += W[j] * dataPre[index[j]];
        dataPost[x + m_nbX * y + nbXY * z] = (unsigned char)val;
    }
}

#endif
