#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * This file is part of the ustk software.
 * Copyright (C) 2016 - 2017 by Inria. All rights reserved.
 *
 * This software is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * ("GPL") version 2 as published by the Free Software Foundation.
 * See the file LICENSE.txt at the root directory of this source
 * distribution for additional information about the GNU GPL.
 *
 * For using ustk with software that can not be combined with the GNU
 * GPL, please contact Inria about acquiring a ViSP Professional
 * Edition License.
 *
 * This software was developed at:
 * Inria Rennes - Bretagne Atlantique
 * Campus Universitaire de Beaulieu
 * 35042 Rennes Cedex
 * France
 *
 * If you have questions regarding the use of this file, please contact
 * Inria at ustk@inria.fr
 *
 * This file is provided AS IS with NO WARRANTY OF ANY KIND, INCLUDING THE
 * WARRANTY OF DESIGN, MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE.
 *
 * Authors:
 * Jason Chevrie
 *
 *****************************************************************************/

#include <hip/hip_runtime.h>

#include <visp3/core/vpException.h>
    

__global__ void kernelPostScanVoxelDirect(unsigned char *dataPost, const unsigned char *dataPre, unsigned int m_nbX, unsigned int m_nbY, unsigned int m_nbZ, int X, int Y, int Z, double m_resolution, double xmax, double ymin, double zmax, unsigned int frameNumber, unsigned int scanLineNumber, double transducerRadius, double motorRadius, double scanLinePitch, double axialResolution, double framePitch, bool sweepInZdirection)
{
    uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
    uint z = (blockIdx.z * blockDim.z) + threadIdx.z;
    
    if(x >= m_nbX  || y >= m_nbY || z >= m_nbZ) return;
  
    unsigned int nbXY = m_nbX * m_nbY;
    unsigned int XY = X * Y;

    double xx = m_resolution * x - xmax;
    double yy = ymin + m_resolution * y;
    double zz = m_resolution * z - zmax;

    double i, j, k;

    double radiusOffset = transducerRadius - motorRadius;
    double rProbe = radiusOffset + sqrt(yy * yy + zz * zz);
    double r = sqrt(rProbe * rProbe + xx * xx);
    double phi = atan(xx / rProbe);
    double theta = atan(zz / yy);

    double itmp = phi / scanLinePitch + 0.5 * (scanLineNumber - 1);
    i = itmp;
    j = (r - transducerRadius) / axialResolution;
    k = (frameNumber * scanLineNumber - 1) * (0.5 / scanLineNumber + theta / (framePitch * frameNumber * scanLineNumber)) - (sweepInZdirection ? itmp : scanLineNumber-1-itmp) / scanLineNumber;
  
    double ii = floor(i);
    double jj = floor(j);
    double kk = floor(k);
  
    if (ii >= 0 && jj >= 0 && kk >= 0 && ii + 1 < X && jj + 1 < Y && kk + 1 < Z) {
              
        double u = i - ii;
        double v = j - jj;
        double w = k - kk;
        double u1 = 1 - u;
        double v1 = 1 - v;
        double w1 = 1 - w;
    
        double v1w1 = v1 * w1;
        double vw1 = v * w1;
        double v1w = v1 * w;
        double vw = v * w;
    
        double W[8] = { u1 * v1w1,
                        u * v1w1,
                        u1 * vw1,
                        u * vw1,
                        u1 * v1w,
                        u * v1w,
                        u1 * vw,
                        u * vw };
        
        double Xjj = X * jj;
        double Xjj1 = X * (jj + 1);
        double XYKK = XY * kk;
        double XYKK1 = XY * (kk + 1);
            
        unsigned int index[8] = { (unsigned int)(ii + Xjj + XYKK),
                                  (unsigned int)(ii + 1 + Xjj + XYKK),
                                  (unsigned int)(ii + Xjj1 + XYKK),
                                  (unsigned int)(ii + 1 + Xjj1 + XYKK),
                                  (unsigned int)(ii + Xjj + XYKK1),
                                  (unsigned int)(ii + 1 + Xjj + XYKK1),
                                  (unsigned int)(ii + Xjj1 + XYKK1),
                                  (unsigned int)(ii + 1 + Xjj1 + XYKK1)};
        
        double val = 0;
        for (int n = 0; n < 8; n++) val += W[n] * dataPre[index[n]];
        dataPost[x + m_nbX * y + nbXY * z] = (unsigned char)val;
    }
}

void GPUDirectConversionWrapper(unsigned char *dataPost, const unsigned char *dataPre, unsigned int m_nbX, unsigned int m_nbY, unsigned int m_nbZ, int X, int Y, int Z, double m_resolution, double xmax, double ymin, double zmax, unsigned int frameNumber, unsigned int scanLineNumber, double transducerRadius, double motorRadius, double scanLinePitch, double axialResolution, double framePitch, bool sweepInZdirection)
{   
    unsigned char *dataPostDevice;
    unsigned int sizePost = m_nbX*m_nbY*m_nbZ*sizeof(unsigned char);
    unsigned char *dataPreDevice;
    unsigned int sizePre = X*Y*Z*sizeof(unsigned char);

	hipError_t codePost = hipMalloc((void**)&dataPostDevice, sizePost);
    if(codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversionWrapper: GPU memory allocation error (%d Bytes)", sizePost);
    hipError_t codePre = hipMalloc((void**)&dataPreDevice, sizePre);
    if(codePre != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversionWrapper: GPU memory allocation error (%d Bytes)", sizePost);

    codePost = hipMemset(dataPostDevice, 0, sizePost);
    if(codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversionWrapper: GPU memory set error");
    codePre = hipMemcpy(dataPreDevice, dataPre, sizePre, hipMemcpyHostToDevice);
    if(codePre != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversionWrapper: GPU memory copy error");

    unsigned int *count;
    count = new unsigned int;
    unsigned int *countDevice;
    hipMalloc((void**)&countDevice, sizeof(unsigned int));
    hipMemset(countDevice, 0, sizeof(unsigned int));

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((m_nbX+threadsPerBlock.x-1)/threadsPerBlock.x, (m_nbY+threadsPerBlock.y-1)/threadsPerBlock.y, (m_nbZ+threadsPerBlock.z-1)/threadsPerBlock.z);
std::cout << numBlocks.x << " " << numBlocks.y << " "  << numBlocks.z << std::endl;
    kernelPostScanVoxelDirect<<<numBlocks,threadsPerBlock>>>(dataPostDevice, dataPreDevice, m_nbX, m_nbY, m_nbZ, X, Y, Z, m_resolution, xmax, ymin, zmax, frameNumber, scanLineNumber, transducerRadius, motorRadius, scanLinePitch, axialResolution, framePitch, sweepInZdirection);

    codePost = hipMemcpy(dataPost, dataPostDevice, sizePost, hipMemcpyDeviceToHost);
    if(codePost != hipSuccess) throw vpException(vpException::memoryAllocationError, "usPreScanToPostScan3DConverter::GPUDirectConversionWrapper: GPU memory copy error");
    hipFree(dataPostDevice);
    hipFree(dataPreDevice);
}
